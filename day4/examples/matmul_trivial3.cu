
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <string>
#include <optional>

__global__ void mul(const double* A, const double* B, double* C, size_t N)
{
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    auto j = threadIdx.y + blockIdx.y * blockDim.y;
    double res {};
    if (i < N && j < N)
        for (size_t k = 0ul; k < N; ++k) {
            res += A[N * i + k] * B[N * k + j];
        }
    C[N * i + j] = res;
}

template <class T>
auto malloc_usm(size_t N, std::optional<T> init = std::nullopt) -> T*
{
    T* ans{};
    hipMallocManaged(&ans, N * sizeof(T));
    if (init) { for (size_t i = 0UL; i < N; ++i) ans[i] = *init; }
    return ans;
}
int main(int argc, char* argv[])
{
    const unsigned N = (argc > 1) ? std::stoul(argv[1]) : 2048u;
    auto a = malloc_usm<double>(N * N);
    auto b = malloc_usm<double>(N * N);
    auto c = malloc_usm<double>(N * N);
    for (size_t i = 0ul; i < N * N; ++i) {
        a[i] = 1.1;
        b[i] = 0.89;
    }
    auto t0 = std::chrono::high_resolution_clock::now();
    dim3 ThreadsPerBlock { 16, 16 };
    dim3 NumBlocks { N / ThreadsPerBlock.x, N / ThreadsPerBlock.y };
    mul<<<NumBlocks, ThreadsPerBlock>>>(a, b, c, N);
    hipDeviceSynchronize();
    auto t1 = std::chrono::high_resolution_clock::now();
    auto dt = std::chrono::duration<double>(t1 - t0).count();
    std::cout << (2.0 * N - 1.) * N * N * 1.0e-9 / dt << " GFlops\n";
    hipFree(c);
    hipFree(b);
    hipFree(a);
}
