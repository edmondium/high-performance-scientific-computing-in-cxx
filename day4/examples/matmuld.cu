#include "hip/hip_runtime.h"
#include <chrono>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>

using namespace thrust::placeholders;

void GPU_fill_random(double* x, int sz)
{
    hiprandGenerator_t cudagen;
    hiprandCreateGenerator(&cudagen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(cudagen, (unsigned long long)clock());
    hiprandGenerateUniformDouble(cudagen, x, sz * sz);
}
void gpu_blas_mmul(const hipblasHandle_t& handle, const double* A, const double* B, double* C, const int sz)
{
    int lda = sz, ldb = sz, ldc = sz;
    const double aa = 1;
    const double bb = 0.;
    const double* alpha = &aa;
    const double* beta = &bb;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sz, sz, sz, alpha, A, lda, B, ldb, beta, C, ldc);
}

int main(int argc, char* argv[])
{
    if (argc < 2) {
        std::cerr << "Usage: \n"
                  << argv[0] << " square_matrix_size repeat_count\n";
        return 1;
    }
    int size = std::stoi(argv[1]);
    int nrep = 10;
    if (argc > 2)
        nrep = std::stoi(argv[2]);
    thrust::device_vector<double> d_A(size * size), d_B(size * size), d_C(size * size);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Fill arrays on the GPU using random number generation on the GPU
    GPU_fill_random(thrust::raw_pointer_cast(&d_A[0]), size);
    GPU_fill_random(thrust::raw_pointer_cast(&d_B[0]), size);
    auto t0 = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < nrep; ++i) {
        gpu_blas_mmul(handle, thrust::raw_pointer_cast(&d_A[0]),
            thrust::raw_pointer_cast(&d_B[0]),
            thrust::raw_pointer_cast(&d_C[0]),
            size);
        hipDeviceSynchronize();
    }
    auto t1 = std::chrono::high_resolution_clock::now();
    auto tdiff = std::chrono::duration<double>(t1 - t0).count();
    std::cout << "Timing for " << nrep << " matrix multiplications on the GPU (measured on the CPU) = "
              << tdiff << "\n";
    std::cout << "Approximately " << (static_cast<size_t>(size) * size * (size + size - 1) * nrep) / (tdiff * 1.0e9) << " GFlops\n";
    hipblasDestroy(handle);
}
