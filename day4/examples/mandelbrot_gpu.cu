#include "hip/hip_runtime.h"
#include "pngwriter.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

using thrust::complex;
using image_representation = typename thrust::host_vector<unsigned char>;
using CI = thrust::counting_iterator<size_t>;

void save_png(std::string ofile, size_t width, size_t height, const image_representation& img)
{
    pngwriter output { static_cast<int>(width), static_cast<int>(height), 0.0, ofile.c_str() };
    for (auto i = 0UL; i < height; ++i) {
        for (auto j = 0UL; j < width; ++j) {
            auto c = img[i * width + j];
            output.plot(j + 1UL, i + 1UL, c / 255.0, c / 255.0,
			    (static_cast<unsigned long>(c) + 50UL) / 255.0);
        }
    }
    output.close();
}

auto mandel(size_t width, size_t height) -> image_representation
{
    thrust::device_vector<unsigned char> res(width * height, 0);
    double aspect = static_cast<double>(width) / height;
    CI beg { 0UL };
    CI end { beg + width * height };
    thrust::transform(beg, end,
        res.begin(), [=] __host__ __device__(size_t index) {
            double myrow = index / width;
            double mycol = index % width;
            myrow /= height;
            mycol /= width;
            complex<double> z0 { aspect * (2 * mycol - 1) - 0.5,
                2 * myrow - 1 };
            unsigned char lim = 255;
            double mag = 0;
            decltype(z0) z{};
            do {
                z = z * z + z0;
                mag = norm(z);
            } while (lim-- && mag < 4.0);
            return lim;
        });
    thrust::host_vector<unsigned char> ans = res;
    return ans;
}

auto main(int argc, char* argv[]) -> int
{
    namespace sc = std::chrono;
    if (argc != 3) {
        std::cerr << "Usage:\n"
                  << argv[0] << " width height\n";
        return 1;
    }
    auto width = std::stoul(argv[1]);
    auto height = std::stoul(argv[2]);
    auto t0 = sc::high_resolution_clock::now();
    auto img = mandel(width, height);
    auto t1 = sc::high_resolution_clock::now();
    std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << sc::duration<double>(t1 - t0).count() << " seconds\n";
    save_png("output.png", width, height, img);
}
