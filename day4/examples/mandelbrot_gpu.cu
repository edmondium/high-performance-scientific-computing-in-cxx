#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <span>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

using thrust::complex;
using image_representation = typename thrust::host_vector<unsigned char>;
using CI = thrust::counting_iterator<size_t>;

void save_pgm(std::string filename, size_t width, size_t height,
    std::span<unsigned char> data)
{
    std::ofstream fout { filename };
    fout << "P2\n"
         << width << " " << height << " 255\n";
    for (size_t row = 0; row < height; ++row) {
        for (size_t col = 0; col < width; ++col) {
            fout << (col ? " " : "")
                 << static_cast<unsigned>(data[row * width + col]);
        }
        fout << "\n";
    }
    fout.close();
}

auto mandel(size_t width, size_t height) -> image_representation
{
    thrust::device_vector<unsigned char> res(width * height, 0);
    double aspect = static_cast<double>(width) / height;
    CI beg { 0UL };
    CI end { beg + width * height };
    thrust::transform(beg, end,
        res.begin(), [=] __host__ __device__(size_t index) {
            double myrow = index / width;
            double mycol = index % width;
            myrow /= height;
            mycol /= width;
            complex<double> z0 { aspect * (2 * mycol - 1) - 0.5,
                2 * myrow - 1 };
            unsigned char lim = 255;
            double mag = 0;
            decltype(z0) z{};
            do {
                z = z * z + z0;
                mag = norm(z);
            } while (lim-- && mag < 4.0);
            return lim;
        });
    thrust::host_vector<unsigned char> ans = res;
    return ans;
}

auto main(int argc, char* argv[]) -> int
{
    namespace sc = std::chrono;
    if (argc != 3) {
        std::cerr << "Usage:\n"
                  << argv[0] << " width height\n";
        return 1;
    }
    auto width = std::stoul(argv[1]);
    auto height = std::stoul(argv[2]);
    auto t0 = sc::high_resolution_clock::now();
    auto img = mandel(width, height);
    auto t1 = sc::high_resolution_clock::now();
    std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << sc::duration<double>(t1 - t0).count() << " seconds\n";
    save_pgm("output.pgm", width, height, std::span(img.data(), img.size()));
}
