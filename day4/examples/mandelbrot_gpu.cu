#include "hip/hip_runtime.h"
#include "pngwriter.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

using thrust::complex;
using image_representation = typename thrust::host_vector<unsigned char>;

void save_png(std::string ofile, int width, int height, const image_representation& img)
{
    pngwriter output { width, height, 0.0, ofile.c_str() };
    for (auto i = 1; i <= height; ++i) {
        for (auto j = 1; j <= width; ++j) {
            auto c = img[i * width + j];
            output.plot(j, i, c / 255.0, c / 255.0, (c + 50) / 255.0);
        }
    }
    output.close();
}

image_representation mandel(size_t width, size_t height)
{
    thrust::device_vector<unsigned char> res(width * height, 0);
    double aspect = static_cast<double>(width) / height;
    thrust::counting_iterator<size_t> indexes{0UL};
    thrust::transform(indexes, indexes + width * height,
        res.begin(), [=] __host__ __device__(size_t index) {
            double myrow = index / width;
            double mycol = index % width;
            myrow /= height;
            mycol /= width;
            thrust::complex<double> z0 { aspect * (2 * mycol - 1) - 0.5,
                2 * myrow - 1 };
            unsigned char lim = 255;
            double mag = 0;
            decltype(z0) z;
            do {
                z = z * z + z0;
                mag = norm(z);
            } while (lim-- && mag < 4.0);
            return lim;
        });
    thrust::host_vector<unsigned char> ans = res;
    return ans;
}

int main(int argc, char* argv[])
{
    if (argc != 3) {
        std::cerr << "Usage:\n"
                  << argv[0] << " width height\n";
        return 1;
    }
    int width = std::stoi(argv[1]);
    int height = std::stoi(argv[2]);
    auto t0 = std::chrono::high_resolution_clock::now();
    auto img = mandel(width, height);
    auto t1 = std::chrono::high_resolution_clock::now();
    std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << std::chrono::duration<double>(t1 - t0).count() << " seconds\n";
    save_png("output.png", width, height, img);
}
