
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <chrono>

__global__ void mul(const double *A, const double *B, double *C, size_t N)
{
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    auto j = threadIdx.y + blockIdx.y * blockDim.y;
    double res{};
    if (i<N && j<N) 
        for (size_t k=0ul; k<N; ++k) {
            res += A[N*i + k] * B[N*k +j];
    }
    C[N*i + j] = res;
}

int main(int argc, char *argv[])
{
    const unsigned N = (argc > 1) ? std::stoul(argv[1]) : 2048u;
    double *a=nullptr, *b=nullptr, *c=nullptr;
    hipMallocManaged(&a, N*N*sizeof(double));
    hipMallocManaged(&b, N*N*sizeof(double));
    hipMallocManaged(&c, N*N*sizeof(double));
    for (size_t i=0ul; i < N*N; ++i) {
        a[i] = 1.1;
        b[i] = 0.89;
    }
    auto t0 = std::chrono::high_resolution_clock::now();
    dim3 ThreadsPerBlock{16,16};
    dim3 NumBlocks{N/ThreadsPerBlock.x,N/ThreadsPerBlock.y};
    mul<<<NumBlocks, ThreadsPerBlock >>>(a, b, c, N);
    hipDeviceSynchronize();
    auto t1 = std::chrono::high_resolution_clock::now();
    auto dt = std::chrono::duration<double>(t1-t0).count();
    std::cout << (2.0 * N -1.) * N * N * 1.0e-9 / dt << " GFlops\n";
}

