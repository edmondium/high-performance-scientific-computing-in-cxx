#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <random>
#include <chrono>

using namespace thrust::placeholders;

int main()
{
    std::random_device seed;
    std::mt19937_64 engine{seed()};
    std::uniform_real_distribution<double> dist;
    auto gen = [&]{ return dist(engine); };
    thrust::host_vector<double> X(1 << 16), Y(1 << 16);
    thrust::generate(X.begin(), X.end(), gen);
    thrust::device_vector<double> dX = X;
    //auto norm = thrust::transform_reduce(dX.begin(), dX.end(), (_1 * _1), 0., (_1 + _2));
    auto norm = thrust::transform_reduce(dX.begin(), dX.end(), 
                                         [] __device__ (auto a){return a*a;}, 
                                         0., 
                                         [] __device__ (auto a, auto b){return a+b;});

    std::cout << "Transform reduce: norm = " << norm << "\n";
}


